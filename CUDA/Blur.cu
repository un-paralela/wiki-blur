#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void blur(const int *image, int *newImage,int ROWS, int COLS, int kernel, int numElements){
    // Calculates the index of the, this index will go from 0 to the number of rows * number of cols
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    // Validates that the calculated index will not be bigger than the size of the image
    if (index < numElements){
        int ch;
        // Iterates over the three channels
		for(ch = 0; ch<3; ch++){
			int sum = 0;
		    int count = 0;
            // Calculates the down limit row
			int limitDownRow = ROWS * COLS * (ch + 1);
            // Calculates the up limit row
		    int limitUpRow = ROWS * COLS * ch - 1;
            // Calculates the 'real' row
		    int currentRow = index / COLS;
            // Calculates the left limit column
		    int limitLeftCol = (currentRow * COLS - 1) + (ROWS * COLS * ch);
            // Calculates the right limit column
		    int limitRightCol = (currentRow + 1) * COLS+ (ROWS * COLS * ch);
            // Calculates the real index of the image vector
			int realIndex = index + (ROWS * COLS * ch);
            int j;
            // The sum of rows or cols will go from -kernel to +kernel
		    for(j=-kernel; j<=kernel; j++){
                // Calculates the new row adding (or subtracting) jth number of cols
		        int newRow = realIndex + j * COLS;
                // The pixel value will be added if it is not the same row, bigger than the lower row limit or lower than the upper row limit
		        if(newRow!=realIndex && newRow>limitUpRow && newRow<limitDownRow){
		            sum+=image[newRow];
		            count++;
		        }
                // Calculates the new column adding (or subtracting) 'j'
		        int newCol = realIndex + j;
                // The pixel value will be added if it is not the same column, bigger than the right columnt limit or lower than the lower row limit
		        if(newCol!=realIndex && newCol>limitLeftCol && newCol<limitRightCol){
		            sum+=image[newCol];
		            count++;
		        }
		    }
            // The blured pixel value will be the division between the added pixel values and the number of added pixel values
		    newImage[realIndex]=sum / count;
		}
    }
}

int readFile(char *fileName, int *array, int indexVec){
    FILE *file;
    int character;
    // Opens the given file
    file = fopen(fileName,"r");
    if (file == NULL){
        fprintf(stderr,"Error de apertura del file");
        exit(EXIT_FAILURE);
    } else {
        // Char array to store a pixel value, it could go from 0 to 255, i.e. 000 to 255
        char num[3] = {};
        int d;
        int index = 0;
        // Reads the file character per character
        while((character = fgetc(file)) != EOF){
            // When the character is different from a tab or a new line, the given value is stored in the char array in the next available position
            if (character != '\t' && character != '\n'){
                num[index++] = character;
            } else {
                // When the character is a tab or a new line it means that it will come a new pixel value
                index = 0;
                // Converts the given char array to an integer value and stores it in the variable 'd'
                sscanf(num, "%d", &d);
                // Stores the las value in the array and increases the indexVec
                array[indexVec++] = d;
                // Empties the char array 'num'
                memset(num,0,sizeof(num));
            }
        }
        // Converts the last given char array to an integer value and stores it in the variable 'd'
        sscanf(num, "%d", &d);
        // Stores the las value in the array
        array[indexVec] = d;
    }
    fclose(file);
    // Returns the last stored position of the array, indicating that the next vector should start at this position
    return indexVec;
}

int main(int argc, char** argv){
	if(argc!=8){
        fprintf(stderr, "The format should be: fileRed fileGreen fileBlue #_Rows #_Cols kernel num_threads");
        exit(EXIT_FAILURE);
    }
    // Reads the number of rows and cols of an image and the given files
    int NUM_ROWS = atoi(argv[4]);
    int NUM_COLS = atoi(argv[5]);
    int NUM_CHS = 3;
    // Checks if the given kernel is odd and bigger than 1
    if(atoi(argv[6])%2==0 && atoi(argv[6])<=1){
        fprintf(stderr, "The number of kernels should be odd and bigger than 1");
        exit(EXIT_FAILURE);
    }
    int kernel = atoi(argv[6])/2;
    // The size of the image vector depends on the rows, cols, channels and the vector will have only int values
    int SIZE = NUM_ROWS * NUM_COLS * NUM_CHS * sizeof(int);
    // Allocate the host image vector
	int *h_image = (int *)malloc(SIZE);
    if(h_image == NULL){
        fprintf(stderr, "Failet do allocate host vector image");
        exit(EXIT_FAILURE);
    }
    // Reads the given file and puts it in the given array starting at the given position
    // For this case, reads the file in the argument 1, stores it in the host image starting in the position 0
    int index = readFile(argv[1],h_image,0);
    index = readFile(argv[2],h_image,index);
    index = readFile(argv[3],h_image,index);
    // Allocate the host newImage vector
	int *h_newImage = (int *)malloc(SIZE);
    if(h_newImage == NULL){
        fprintf(stderr, "Failet do allocate host vector newImage");
        exit(EXIT_FAILURE);
    }
	hipError_t err = hipSuccess;
	int *d_image = NULL;
    // Allocate the device image vector
	err = hipMalloc((void **)&d_image, SIZE);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Allocate the device newImage vector
    int *d_newImage = NULL;
	err = hipMalloc((void **)&d_newImage, SIZE);
	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector newImage (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copies the memory from the host image to the device image
    err = hipMemcpy(d_image, h_image, SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy first dimesion of image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // The threads per block will depend on a given argument of the user
    int threadsPerBlock = atoi(argv[7]);
    int totalThreads = NUM_ROWS * NUM_COLS + threadsPerBlock - 1;
    // Calculates the number of blocks per grid that will depend on the number of rows and cols of the image
    int blocksPerGrid = totalThreads / threadsPerBlock;
    // Launches the blur function with the device image, the device newImage (result image), number of rows, cols, number of kernel and the total elements, i.e. size of each channel
    // Each thread will be responsible of calculate the blur effect on an especific pixel in their three channels
    blur<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_newImage,NUM_ROWS, NUM_COLS, kernel, NUM_ROWS * NUM_COLS);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch blur kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Copies the memory from the device newImage to the host newImage
    err = hipMemcpy(h_newImage, d_newImage, SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy first dimesion of image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Creates three files, in which will be stored the result of the blur, each file will be a channel of the blured image
	const char *files[3] = {"redResult.txt","greenResult.txt","blueResult.txt"};
	int ch;
	for(ch = 0; ch < NUM_CHS; ch++){
		FILE *file = fopen(files[ch],"w");
		for(index = NUM_ROWS * NUM_COLS * ch; index < NUM_ROWS * NUM_COLS * (ch + 1); index++){
			fprintf(file, "%i\t", h_newImage[index]);
		}
		fclose(file);
	}
    // Frees the memory of the device image
    err = hipFree(d_image);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Frees the memory of the device newImage
    err = hipFree(d_newImage);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector newImage (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // Frees the memory of the host image
    free(h_image);
    // Frees the memory of the host newImage
    free(h_newImage);
}